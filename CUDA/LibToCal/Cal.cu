#include "hip/hip_runtime.h"
#include "Cal.cuh"
#include <stdio.h> 
#include<math.h>

const int ThreadCount = 1024;
const int MaxValue = 60000 * 1000;

__global__ void getMinStepFF(int length, int segCountPerUnit, int pointCountPerUnit, int* startDic, int* endDic, int* lastFP, int* minStepResult, int* minStepResultOnOff, int* costTime)
{
	//int bx = blockIdx.x;

	//int tx = threadIdx.x;
	//getMinStepF << <this->BlockCount, ThreadCount >> > (this->Length, this->LeftStepWithPass_GPU, this->MinStepResult_GPU);
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < length) {

		//if(i>=input.)
		int startIndex = startDic[i % segCountPerUnit];
		int endIndex = endDic[i % segCountPerUnit];
		int unitIndex = i / segCountPerUnit;

		//int segIndex=

		if (lastFP[endIndex + unitIndex * pointCountPerUnit] == -1)
		{
			if (lastFP[startIndex + unitIndex * pointCountPerUnit] == -1)
			{
				minStepResult[i] = MaxValue;
				minStepResultOnOff[i] = 0;
			}
			else
			{
				minStepResult[i] = costTime[i];
				minStepResultOnOff[i] = 1;
			}
		}
		else
		{
			minStepResult[i] = MaxValue;
			minStepResultOnOff[i] = 0;
		}
	}
}


__global__ void getReduceF(int length, int segCountPerUnit, int pointCountPerUnit, int* startDic, int* endDic, int* lastFP, int* minStepResult, int* minStepResultOnOff, int* costTime)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < length) {
		int startIndex = startDic[i % segCountPerUnit];
		int endIndex = endDic[i % segCountPerUnit];
		int unitIndex = i / segCountPerUnit;
		if (minStepResultOnOff[i] == 1)
		{
			if (costTime[i] - minStepResult[i] * minStepResultOnOff[i] == 0)
			{
				if (lastFP[unitIndex * pointCountPerUnit + endIndex] == -1)
				{
					lastFP[unitIndex * pointCountPerUnit + endIndex] = startIndex;
				}

			}
			else
			{

			}
			costTime[i] -= minStepResult[i] * minStepResultOnOff[i];
		}
	}
}

__global__ void FindMinOfMinStepResult(int length, int segCountPerUnit, int step, int* minStepResult)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < length)
	{
		int indexOfItem = i % segCountPerUnit;
		int indexOfUnit = i / segCountPerUnit;
		int indexBaseOfUnit = indexOfUnit * segCountPerUnit;



		if ((i - indexBaseOfUnit) % (step << 1) == 0)

			if (i - indexBaseOfUnit < segCountPerUnit) {
				if (i - indexBaseOfUnit + step < indexBaseOfUnit + segCountPerUnit)
				{
					minStepResult[i] =
						minStepResult[i] > minStepResult[i + step] ? minStepResult[i] : minStepResult[i + step];
				}
			}
	}
}


__global__ void CopyMinStepResult(int length, int* minStepResult, int* minStepResultOut)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < length)
	{
		minStepResultOut[i] = minStepResult[i];
	}
}
Cal::Cal(int* costTime, int* lastFP, int* resultForSave, int costTimeCount, int fPCount, int calUnitCount, int* startDic, int* endDic)
{

	this->CostTimeCount = costTimeCount;
	this->PointCount = fPCount;
	this->UnitCount = calUnitCount;
	this->PointCountPerUnit = fPCount / calUnitCount;
	this->SegCountPerUnit = costTimeCount / calUnitCount;

	if (this->CostTimeCount % ThreadCount == 0)
	{
		this->BlockCount = this->CostTimeCount / ThreadCount;
	}
	else
	{
		this->BlockCount = this->CostTimeCount / ThreadCount + 1;
	}



	//this->MinStepResult = minStepResult;
	//this->MinStepValue = MaxValue;

	//this->Direct_GPU = 0;
	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "getReduceF launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
	{
		this->CostTime = costTime;


		this->CostTime_GPU = 0;
		cudaStatus = hipMalloc((void**)&this->CostTime_GPU, this->CostTimeCount * sizeof(int));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "getReduceF launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}
		cudaStatus = hipMemcpy(this->CostTime_GPU, this->CostTime, this->CostTimeCount * sizeof(int), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "getReduceF launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}
	}
	{
		this->LastFP = lastFP;
		this->LastFP_GPU = 0;
		this->LastFP_Out_GPU = 0;

		cudaStatus = hipMalloc((void**)&this->LastFP_GPU, this->PointCount * sizeof(int));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "getReduceF launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}
		cudaStatus = hipMemcpy(this->LastFP_GPU, this->CostTime, this->CostTimeCount * sizeof(int), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "getReduceF launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}
	}
	{
		this->StartDic = startDic;
		this->StartDic_GPU = 0;

		cudaStatus = hipMalloc((void**)&this->StartDic_GPU, this->PointCount / this->UnitCount * sizeof(int));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "getReduceF launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}
		cudaStatus = hipMemcpy((void**)&this->StartDic_GPU, this->StartDic, this->PointCount / this->UnitCount * sizeof(int), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "getReduceF launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}
	}

	{
		this->EndDic = endDic;
		this->EndDic_GPU = 0;

		cudaStatus = hipMalloc((void**)&this->EndDic_GPU, this->PointCount / this->UnitCount * sizeof(int));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "getReduceF launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}
		cudaStatus = hipMemcpy((void**)&this->EndDic_GPU, this->EndDic, this->PointCount / this->UnitCount * sizeof(int), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "getReduceF launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}
	}
	{
		this->MinStepResult_GPU = 0;

		cudaStatus = hipMalloc((void**)&this->MinStepResult_GPU, this->CostTimeCount * sizeof(int));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "getReduceF launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}
		cudaStatus = hipMemcpy(this->MinStepResult_GPU, this->CostTime, this->CostTimeCount * sizeof(int), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "getReduceF launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}
	}
	//MinStepResult_OnOff_GPU

	{
		this->MinStepResult_OnOff_GPU = 0;

		cudaStatus = hipMalloc((void**)&this->MinStepResult_OnOff_GPU, this->CostTimeCount * sizeof(int));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "getReduceF launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}
		cudaStatus = hipMemcpy(this->MinStepResult_OnOff_GPU, this->CostTime, this->CostTimeCount * sizeof(int), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "getReduceF launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}
	}

	do {
		CalculateMinStep();
		Reduce();
		Copy();
	} while (NotFinished());

	cudaStatus = hipMemcpy(this->LastRecord, this->LastRecord_GPU, this->Length * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "copy Result error!\n", cudaStatus);
		goto Error;
	}
	hipFree(this->Direct_GPU);
	hipFree(this->IsTargetFP_GPU);
	hipFree(this->LastRecord_GPU);
	hipFree(this->LastRecord_Out_GPU);
	hipFree(this->LeftStepWithPass_GPU);
	hipFree(this->LeftStepWithPass_Out_GPU);
	hipFree(this->MinStepResult_GPU);
	hipFree(this->NotPassedTargetFPStaticc_GPU);
	return;
Error:
	/*hipFree(LeftStep_Input);
	hipFree(MinStep_Output);*/
	hipFree(this->Direct_GPU);
	hipFree(this->IsTargetFP_GPU);
	hipFree(this->LastRecord_GPU);
	hipFree(this->LastRecord_Out_GPU);
	hipFree(this->LeftStepWithPass_GPU);
	hipFree(this->LeftStepWithPass_Out_GPU);
	hipFree(this->MinStepResult_GPU);
	hipFree(this->NotPassedTargetFPStaticc_GPU);
	return;
}



hipError_t Cal::CalculateMinStep()
{
	hipError_t cudaStatus;
	// this->CostTime_GPU
			// Launch a kernel on the GPU with one thread for each element.
	getMinStepFF << <this->BlockCount, ThreadCount >> > (this->CostTimeCount, this->SegCountPerUnit, this->PointCountPerUnit, this->StartDic_GPU, this->EndDic_GPU, this->LastFP_GPU, this->MinStepResult_GPU, this->CostTime_GPU);
	//addKernel(dev_c, dev_a, dev_b);
	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	//If you look in the programming guide you will see that the maximum amount of threads per block is 512
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "getMinStepFF launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	FindMin();
	return cudaStatus;
Error:
	/*hipFree(LeftStep_Input);
	hipFree(MinStep_Output);*/
	return cudaStatus;
}

hipError_t Cal::FindMin()
{
	hipError_t cudaStatus;
	int step = 1;
	while (step < this->SegCountPerUnit)
	{

		FindMinOfMinStepResult << <this->BlockCount, ThreadCount >> > (this->CostTimeCount, this->SegCountPerUnit, step, this->MinStepResult_GPU);
		//hipError_t cudaStatus;
		cudaStatus = hipGetLastError();
		//If you look in the programming guide you will see that the maximum amount of threads per block is 512
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "FindMinOfMinStepResult launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}
		/*	cudaStatus = hipDeviceSynchronize();
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
				goto Error;
			}*/
		step = step << 1;
	}
	return cudaStatus;
Error:
	/*hipFree(LeftStep_Input);
	hipFree(MinStep_Output);*/
	return cudaStatus;
	//return hipError_t();
}

hipError_t Cal::Reduce()
{
	//	this->MinStepResult_GPU
	hipError_t cudaStatus;

	getReduceF << <this->BlockCount, ThreadCount >> > (this->Length, this->MinStepResult_GPU, this->LeftStepWithPass_GPU, this->LeftStepWithPass_Out_GPU, this->LastRecord_GPU, this->LastRecord_Out_GPU, this->Direct_GPU);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "getReduceF launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}
	return cudaStatus;
Error:
	/*hipFree(LeftStep_Input);
	hipFree(MinStep_Output);*/
	return cudaStatus;
}

hipError_t Copy()
{
	hipError_t cudaStatus;
	{
		CopyMinStepResult << <this->BlockCount, ThreadCount >> > (this->Length, this->LeftStepWithPass_GPU, this->LeftStepWithPass_Out_GPU);
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "CopyLeftStepWithPass launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
			goto Error;
		}
	} 
	return cudaStatus;
Error:
	return cudaStatus;
}