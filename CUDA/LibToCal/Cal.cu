#include "hip/hip_runtime.h"
#include "Cal.cuh"
#include <stdio.h> 
#include<math.h>

const int ThreadCount = 1024;
const int MaxValue = 60000 * 1000;

/// <summary>
/// ��ȡ��С��ǰ��ֵ��
/// </summary>
/// <param name="length">CostTimeCount</param>
/// <param name="segCountPerUnit"></param>
/// <param name="pointCountPerUnit"></param>
/// <param name="startDic"></param>
/// <param name="endDic"></param>
/// <param name="lastFP"></param>
/// <param name="minStepResult"></param>
/// <param name="minStepResultOnOff"></param>
/// <param name="costTime"></param>
/// <returns></returns>
__global__ void getMinStepFF(int length, int segCountPerUnit, int pointCountPerUnit, int* startDic, int* endDic, int* lastFP, int* minStepResult, int* minStepResultOnOff, int* costTime)
{
	//int bx = blockIdx.x;

	//int tx = threadIdx.x;
	//getMinStepF << <this->BlockCount, ThreadCount >> > (this->Length, this->LeftStepWithPass_GPU, this->MinStepResult_GPU);
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < length) {

		//if(i>=input.)
		int startIndex = startDic[i % segCountPerUnit];
		int endIndex = endDic[i % segCountPerUnit];
		int unitIndex = i / segCountPerUnit;

		//int segIndex=

		if (lastFP[endIndex + unitIndex * pointCountPerUnit] == -1)
		{
			if (lastFP[startIndex + unitIndex * pointCountPerUnit] == -1)
			{
				minStepResult[i] = MaxValue;
				minStepResultOnOff[i] = 0;
			}
			else
			{
				minStepResult[i] = costTime[i];
				minStepResultOnOff[i] = 1;
			}
		}
		else
		{
			minStepResult[i] = MaxValue;
			minStepResultOnOff[i] = 0;
		}
	}
}


__global__ void getReduceF(int length, int segCountPerUnit, int pointCountPerUnit, int* startDic, int* endDic, int* lastFP, int* minStepResult, int* minStepResultOnOff, int* costTime)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	//int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < length) {
		int startIndex = startDic[i % segCountPerUnit];
		int endIndex = endDic[i % segCountPerUnit];
		int unitIndex = i / segCountPerUnit;
		if (minStepResultOnOff[i] == 1)
		{
			if (costTime[i] - minStepResult[i] * minStepResultOnOff[i] == 0)
			{
				if (lastFP[unitIndex * pointCountPerUnit + endIndex] == -1)
				{
					lastFP[unitIndex * pointCountPerUnit + endIndex] = startIndex;
				}

			}
			else
			{

			}
			costTime[i] -= minStepResult[i] * minStepResultOnOff[i];
		}
	}
}

__global__ void FindMinOfMinTimeStepResult(int length, int segCountPerUnit, int step, int* minStepResult)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < length)
	{
		int columnIndex = i % segCountPerUnit;
		if (columnIndex % (step << 1) == 0) {
			if (columnIndex + step < segCountPerUnit)
			{
				minStepResult[i] =
					minStepResult[i] < minStepResult[i + step] ? minStepResult[i] : minStepResult[i + step];
			}
		}
	}
}


__global__ void CopyMinStepResult(int length, int* minStepResult, int* minStepResultOut)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < length)
	{
		minStepResultOut[i] = minStepResult[i];
	}
}

__global__ void FindMinOfIndexOfFPStepResult(int length, int step, int pointCountPerUnit, int* minStepResultOut)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < length) {
		int columnIndex = i % pointCountPerUnit;
		if (columnIndex % (step << 1) == 0)
		{
			if (columnIndex + step < pointCountPerUnit)
			{
				minStepResultOut[i] =
					minStepResultOut[i] < minStepResultOut[i + step] ? minStepResultOut[i] : minStepResultOut[i + step];
			}
		}
	}
}

Cal::Cal(int* costTime, int* lastFP, int costTimeCount, int fPCount, int calUnitCount, int* startDic, int* endDic)
{

	this->CostTimeCount = costTimeCount;
	this->PointCount = fPCount;
	this->UnitCount = calUnitCount;
	this->PointCountPerUnit = fPCount / calUnitCount;
	this->SegCountPerUnit = costTimeCount / calUnitCount;

	if (this->CostTimeCount % ThreadCount == 0)
	{
		this->BlockCount = this->CostTimeCount / ThreadCount;
	}
	else
	{
		this->BlockCount = this->CostTimeCount / ThreadCount + 1;
	}



	//this->MinStepResult = minStepResult;
	//this->MinStepValue = MaxValue;

	//this->Direct_GPU = 0;

	hipError_t cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "161cudaDeviceReset failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	/*cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "167--started failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}*/
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}
	{
		//hipError_t cudaStatus = hipDeviceSynchronize();
		//if (cudaStatus != hipSuccess) {
		//	fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		//	//goto Error;
		//}
	}
	{
		this->CostTime = costTime;


		this->CostTime_GPU = 0;
		cudaStatus = hipMalloc((void**)&this->CostTime_GPU, this->CostTimeCount * sizeof(int));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "CostTime_GPU hipMalloc failed!\n");
			goto Error;
		}
		/*	cudaStatus = hipGetLastError();
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "176--CostTime_GPU launch failed: %s\n", hipGetErrorString(cudaStatus));
				goto Error;
			}*/
		cudaStatus = hipMemcpy(this->CostTime_GPU, this->CostTime, this->CostTimeCount * sizeof(int), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "CostTime_GPU hipMemcpy failed!\n");
			goto Error;
		}
		/*	cudaStatus = hipGetLastError();
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "CostTime_GPU hipMemcpy failed!%s\n", hipGetErrorString(cudaStatus));
				goto Error;
			}*/
	}
	{
		this->LastFP = lastFP;
		this->LastFP_GPU = 0;
		this->LastFP_Out_GPU = 0;

		cudaStatus = hipMalloc((void**)&this->LastFP_GPU, this->PointCount * sizeof(int));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "LastFP_GPU hipMalloc failed!");
			goto Error;
		}
		/*cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "LastFP_GPU launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}*/
		cudaStatus = hipMemcpy(this->LastFP_GPU, this->LastFP, this->PointCount * sizeof(int), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "LastFP_GPU failed!");
			goto Error;
		}
		/*cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "LastFP_GPU copy failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}*/

		cudaStatus = hipMalloc((void**)&this->LastFP_Out_GPU, this->PointCount * sizeof(int));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}
		/*cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "LastFP_GPU launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}*/

	}
	{
		this->StartDic = startDic;
		this->StartDic_GPU = 0;

		cudaStatus = hipMalloc((void**)&this->StartDic_GPU, this->SegCountPerUnit * sizeof(int));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}
		/*cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "StartDic_GPU launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}*/
		cudaStatus = hipMemcpy(this->StartDic_GPU, this->StartDic, this->SegCountPerUnit * sizeof(int), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}
		/*cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "StartDic_GPU launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}*/
	}

	{
		this->EndDic = endDic;
		this->EndDic_GPU = 0;

		cudaStatus = hipMalloc((void**)&this->EndDic_GPU, this->SegCountPerUnit * sizeof(int));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}
		/*cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "EndDic_GPU launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}*/
		cudaStatus = hipMemcpy(this->EndDic_GPU, this->EndDic, this->SegCountPerUnit * sizeof(int), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}
		/*cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "EndDic_GPU launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}*/
	}
	{
		this->MinStepResult_GPU = 0;

		cudaStatus = hipMalloc((void**)&this->MinStepResult_GPU, this->CostTimeCount * sizeof(int));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}
		/*	cudaStatus = hipGetLastError();
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "MinStepResult_GPU launch failed: %s\n", hipGetErrorString(cudaStatus));
				goto Error;
			}*/
		cudaStatus = hipMemcpy(this->MinStepResult_GPU, this->CostTime, this->CostTimeCount * sizeof(int), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}
		/*	cudaStatus = hipGetLastError();
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "MinStepResult_GPU launch failed: %s\n", hipGetErrorString(cudaStatus));
				goto Error;
			}*/
	}

	/*{
		this->MinStepResult_CALMINVALUE_GPU = 0;

		cudaStatus = hipMalloc((void**)&this->MinStepResult_CALMINVALUE_GPU, this->CostTimeCount * sizeof(int));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "getReduceF launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}
		cudaStatus = hipMemcpy(this->MinStepResult_CALMINVALUE_GPU, this->CostTime, this->CostTimeCount * sizeof(int), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "getReduceF launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}
	}*/
	//MinStepResult_CALMINVALUE_GPU
	//MinStepResult_OnOff_GPU

	{
		this->MinStepResult_OnOff_GPU = 0;

		cudaStatus = hipMalloc((void**)&this->MinStepResult_OnOff_GPU, this->CostTimeCount * sizeof(int));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}
		/*cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "MinStepResult_OnOff_GPU launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}*/
		cudaStatus = hipMemcpy(this->MinStepResult_OnOff_GPU, this->CostTime, this->CostTimeCount * sizeof(int), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}
		/*cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "MinStepResult_OnOff_GPU launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}*/
	}

	do {
		CalculateMinStep();
		Reduce();
		Copy();
	} while (NotFinished());

	Copy();


	{
		cudaStatus = hipMemcpy(this->LastFP, this->LastFP_GPU, this->PointCount * sizeof(int), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed 377!");
			goto Error;
		}
	}


	hipFree(this->CostTime_GPU);
	hipFree(this->LastFP_GPU);
	hipFree(this->LastFP_Out_GPU);
	hipFree(this->StartDic_GPU);
	hipFree(this->EndDic_GPU);
	hipFree(this->MinStepResult_GPU);
	hipFree(this->MinStepResult_OnOff_GPU);

	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
	this->LastFPResult = this->LastFP;

	return;
Error:
	hipFree(this->CostTime_GPU);
	hipFree(this->LastFP_GPU);
	hipFree(this->LastFP_Out_GPU);
	hipFree(this->StartDic_GPU);
	hipFree(this->EndDic_GPU);
	hipFree(this->MinStepResult_GPU);
	hipFree(this->MinStepResult_OnOff_GPU);

	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset Error failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	return;
}


//int length, int segCountPerUnit, int pointCountPerUnit, int* startDic, int* endDic, int* lastFP, int* minStepResult, int* minStepResultOnOff, int* costTime
hipError_t Cal::CalculateMinStep()
{
	hipError_t cudaStatus;
	// this->CostTime_GPU
			// Launch a kernel on the GPU with one thread for each element.

	getMinStepFF << <this->BlockCount, ThreadCount >> > (this->CostTimeCount, this->SegCountPerUnit, this->PointCountPerUnit, this->StartDic_GPU, this->EndDic_GPU, this->LastFP_GPU, this->MinStepResult_GPU, this->MinStepResult_OnOff_GPU, this->CostTime_GPU);
	//addKernel(dev_c, dev_a, dev_b);
	// Check for any errors launching the kernel


	cudaStatus = hipGetLastError();
	//If you look in the programming guide you will see that the maximum amount of threads per block is 512
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "getMinStepFF launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	FindMin();
	return cudaStatus;
Error:
	/*hipFree(LeftStep_Input);
	hipFree(MinStep_Output);*/
	return cudaStatus;
}

hipError_t Cal::FindMin()
{
	hipError_t cudaStatus;
	int step = 1;
	while (step < this->SegCountPerUnit)
	{
		//FindMinOfMinTimeStepResult(int length, int segCountPerUnit, int step, int* minStepResult)

		FindMinOfMinTimeStepResult << <this->BlockCount, ThreadCount >> > (this->CostTimeCount, this->SegCountPerUnit, step, this->MinStepResult_GPU);
		//FindMinOfMinStepResult << <this->BlockCount, ThreadCount >> > (this->CostTimeCount, this->SegCountPerUnit, step, this->MinStepResult_GPU);
		//hipError_t cudaStatus;
		cudaStatus = hipGetLastError();
		//If you look in the programming guide you will see that the maximum amount of threads per block is 512
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "FindMinOfMinStepResult launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
			goto Error;
		}
		step = step << 1;
	}
	return cudaStatus;
Error:
	/*hipFree(LeftStep_Input);
	hipFree(MinStep_Output);*/
	return cudaStatus;
	//return hipError_t();
}

hipError_t Cal::Reduce()
{
	//	this->MinStepResult_GPU
	hipError_t cudaStatus;
	//int length, int segCountPerUnit, int pointCountPerUnit, int* startDic, int* endDic, int* lastFP, int* minStepResult, int* minStepResultOnOff, int* costTime
	getReduceF << <this->BlockCount, ThreadCount >> > (
		this->CostTimeCount,
		this->SegCountPerUnit,
		this->PointCountPerUnit,
		this->StartDic_GPU,
		this->EndDic_GPU,
		this->LastFP_GPU,
		this->MinStepResult_GPU,
		this->MinStepResult_OnOff_GPU,
		this->CostTime_GPU
		);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "getReduceFun launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}
	return cudaStatus;
Error:
	/*hipFree(LeftStep_Input);
	hipFree(MinStep_Output);*/
	return cudaStatus;
}

hipError_t  Cal::Copy()
{
	hipError_t cudaStatus;
	{
		CopyMinStepResult << <this->BlockCount, ThreadCount >> > (this->PointCount, this->LastFP_GPU, this->LastFP_Out_GPU);
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "CopyLeftStepWithPass launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
			goto Error;
		}

		//CopyMinStepResult << <this->BlockCount, ThreadCount >> >��this
	}
	return cudaStatus;
Error:
	return cudaStatus;
}

bool Cal::NotFinished()
{
	int step = 1;

	while (step < this->PointCountPerUnit)
	{
		//nt length, int step, int* minStepResultOut
	//	this->MinStepResult_GPU  

		FindMinOfIndexOfFPStepResult << <this->BlockCount, ThreadCount >> > (this->PointCount, step, this->PointCountPerUnit, this->LastFP_Out_GPU);

		hipError_t	cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "MinStepResult_OnOff_GPU launch failed: %s\n", hipGetErrorString(cudaStatus));

		}
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
			//	goto Error;
		}

		step = step << 1;
	}

	//	this->PointCount
	//int h_data[this->PointCount];
	int* h_data = new int[this->PointCount];
	hipError_t cudaStatus = hipMemcpy(h_data, this->LastFP_Out_GPU, this->PointCount * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		//goto Error;
	}
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		//	goto Error;
	}
	for (int i = 0; i < this->PointCount; i++)
	{
		fprintf(stderr, "%d ��", h_data[i]);
	}
	fprintf(stderr, "\n");
	//hipError_t cudaStatus = hipDeviceSynchronize();
	//if (cudaStatus != hipSuccess) {
	//	fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
	//	//goto Error;
	//}
	bool notFinished = false;
	for (int i = 0; i < this->PointCount; i += this->PointCountPerUnit)
	{
		notFinished = notFinished || (h_data[i] == -1);
		if (notFinished)
		{
			break;
		}
	}
	return notFinished;
}

Cal::~Cal()
{
	hipFree(this->CostTime_GPU);
	hipFree(this->LastFP_GPU);
	hipFree(this->LastFP_Out_GPU);
	hipFree(this->StartDic_GPU);
	hipFree(this->EndDic_GPU);
	hipFree(this->MinStepResult_GPU);
	//hipFree(this->MinStepResult_CALMINVALUE_GPU);
	hipFree(this->MinStepResult_OnOff_GPU);
	hipError_t	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
	}
	else
	{
		//fprintf(stderr, "~Cal() hipDeviceReset success!");
	}
}

extern "C" __declspec(dllexport) Cal * MCal_Create(int* costTime, int* lastFP, int costTimeCount, int fPCount, int calUnitCount, int* startDic, int* endDic)
{
	Cal* rc = new Cal(costTime, lastFP, costTimeCount, fPCount, calUnitCount, startDic, endDic);
	return rc;
}

extern "C" __declspec(dllexport) void  MCal_Delete(Cal * cal)
{
	cal->~Cal();
	//pFoo->~PolygonExtrude(); 
	delete cal;
}
extern "C" __declspec(dllexport) int* MCal_LastFPResult(Cal * cal)
{
	return	cal->LastFPResult;
}